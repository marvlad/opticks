#include "hip/hip_runtime.h"
/**
QPMT.cu
==========

_QPMT_lpmtcat_rindex
_QPMT_lpmtcat_qeshape
_QPMT_lpmtcat_stackspec
    kernel funcs taking (qpmt,lookup,domain,domain_width) args

QPMT_pmtcat_scan
    CPU entry point to launch above kernels controlled by etype


_QPMT_lpmtid_stackspec
    kernel funcs taking (qpmt,lookup,domain,domain_width,lpmtid,num_lpmtid) args

_QPMT_mct_lpmtid
    payload size P templated kernel function with domain and lpmtid array inputs

    * within lpmtid loop calls qpmt.h method depending on etype
    * etype : (qpmt_SPEC qpmt_LL qpmt_COMP qpmt_ART qpmt_ARTE)

QPMT_mct_lpmtid_scan
    CPU entry point to launch above kernel passing etype


**/

#include "QUDARAP_API_EXPORT.hh"
#include <stdio.h>
#include "qpmt_enum.h"
#include "qpmt.h"
#include "qprop.h"


/**
_QPMT_lpmtcat_rindex
---------------------------

max_iprop::

   . (ni-1)*nj*nk + (nj-1)*nk + (nk-1)
   =  ni*nj*nk - nj*nk + nj*nk - nk + nk - 1
   =  ni*nj*nk - 1


HMM: not so easy to generalize from rindex to also do qeshape
because of the different array shapes

Each thread does all pmtcat,layers and props for a single energy_eV.

**/

template <typename F>
__global__ void _QPMT_lpmtcat_rindex( int etype, qpmt<F>* pmt, F* lookup , const F* domain, unsigned domain_width )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;
    F domain_value = domain[ix] ;    // energy_eV

    //printf("//_QPMT_rindex domain_width %d ix %d domain_value %10.4f \n", domain_width, ix, domain_value );
    // wierd unsigned/int diff between qpmt.h and here ? to get it to compile for device
    // switching to enum rather than constexpr const avoids the wierdness

    const int& ni = s_pmt::NUM_CAT ;
    const int& nj = s_pmt::NUM_LAYR ;
    const int& nk = s_pmt::NUM_PROP ;

    //printf("//_QPMT_lpmtcat_rindex ni %d nj %d nk %d \n", ni, nj, nk );
    // cf the CPU equivalent NP::combined_interp_5

    for(int i=0 ; i < ni ; i++)
    for(int j=0 ; j < nj ; j++)
    for(int k=0 ; k < nk ; k++)
    {
        int iprop = i*nj*nk+j*nk+k ;            // linearized higher dimensions
        int index = iprop * domain_width + ix ; // output index into lookup

        F value = pmt->rindex_prop->interpolate(iprop, domain_value );

        //printf("//_QPMT_lpmtcat_rindex iprop %d index %d value %10.4f \n", iprop, index, value );

        lookup[index] = value ;
    }
}



template <typename F>
__global__ void _QPMT_lpmtcat_stackspec( int etype, qpmt<F>* pmt, F* lookup , const F* domain, unsigned domain_width )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;
    F domain_value = domain[ix] ;

    //printf("//_QPMT_lpmtcat_stackspec domain_width %d ix %d domain_value %10.4f \n", domain_width, ix, domain_value );

    const int& ni = s_pmt::NUM_CAT ;
    const int& nj = domain_width ;
    const int  nk = 16 ;
    const int&  j = ix ;

    F ss[nk] ;

    for(int i=0 ; i < ni ; i++)  // over pmtcat
    {
        int index = i*nj*nk + j*nk  ;
        pmt->get_lpmtcat_stackspec(ss, i, domain_value );
        for( int k=0 ; k < nk ; k++) lookup[index+k] = ss[k] ;
    }
}



template <typename F>
__global__ void _QPMT_pmtcat_launch( int etype, qpmt<F>* pmt, F* lookup , const F* domain, unsigned domain_width )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;
    F domain_value = domain[ix] ;

    //printf("//_QPMT_pmtcat_launch etype %d domain_width %d ix %d  \n", etype, domain_width, ix  );

    const int ni = ( etype == qpmt_S_QESHAPE ) ? 1 : s_pmt::NUM_CAT ;

    for(int i=0 ; i < ni ; i++)
    {
        int pmtcat = i ;
        F value = 0.f ;

        if( etype == qpmt_QESHAPE )
        {
            value = pmt->qeshape_prop->interpolate( pmtcat, domain_value );
        }
        else if( etype == qpmt_CETHETA )
        {
            //value = pmt->cetheta_prop->interpolate(lpmtcat, domain_value );
            value = pmt->get_lpmtcat_ce( pmtcat, domain_value );
        }
        else if ( etype == qpmt_CECOSTH )
        {
            value = pmt->cecosth_prop->interpolate( pmtcat, domain_value );
        }
        else if( etype == qpmt_S_QESHAPE )
        {
            value = pmt->s_qeshape_prop->interpolate( pmtcat, domain_value );
        }


        int index = i * domain_width + ix ; // output index into lookup
        lookup[index] = value ;
    }
}




/**
QPMT_pmtcat_scan
-------------------

Performs CUDA launches, invoked from QPMT.cc QPMT<T>::pmtcat_scan

**/


template <typename F> extern void QPMT_pmtcat_scan(
    dim3 numBlocks,
    dim3 threadsPerBlock,
    qpmt<F>* pmt,
    int etype,
    F* lookup,
    const F* domain,
    unsigned domain_width
)
{

    switch(etype)
    {
        case qpmt_RINDEX     : _QPMT_lpmtcat_rindex<F><<<numBlocks,threadsPerBlock>>>(    etype, pmt, lookup, domain, domain_width )   ; break ;
        case qpmt_CATSPEC    : _QPMT_lpmtcat_stackspec<F><<<numBlocks,threadsPerBlock>>>( etype, pmt, lookup, domain, domain_width )   ; break ;
        case qpmt_QESHAPE    : _QPMT_pmtcat_launch<F><<<numBlocks,threadsPerBlock>>>(    etype, pmt, lookup, domain, domain_width )   ; break ;
        case qpmt_CETHETA    : _QPMT_pmtcat_launch<F><<<numBlocks,threadsPerBlock>>>(    etype, pmt, lookup, domain, domain_width )   ; break ;
        case qpmt_CECOSTH    : _QPMT_pmtcat_launch<F><<<numBlocks,threadsPerBlock>>>(    etype, pmt, lookup, domain, domain_width )   ; break ;
        case qpmt_S_QESHAPE  : _QPMT_pmtcat_launch<F><<<numBlocks,threadsPerBlock>>>(    etype, pmt, lookup, domain, domain_width )   ; break ;
    }
}

template void QPMT_pmtcat_scan(
   dim3,
   dim3,
   qpmt<float>*,
   int etype,
   float*,
   const float* ,
   unsigned
  );








/**
_QPMT_lpmtid_stackspec
-------------------------

**/


template <typename F>
__global__ void _QPMT_lpmtid_stackspec(
    qpmt<F>* pmt,
    F* lookup ,
    const F* domain,
    unsigned domain_width,
    const int* lpmtid,
    unsigned num_lpmtid )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;
    F energy_eV = domain[ix] ;

    const int& ni = num_lpmtid ;
    const int& nj = domain_width ;
    const int  nk = 16 ;
    const int&  j = ix ;

    F ss[nk] ;

    for(int i=0 ; i < ni ; i++)  // over num_lpmtid
    {
        int pmtid = lpmtid[i] ;
        int index = i*nj*nk + j*nk  ;
        pmt->get_lpmtid_stackspec(ss, pmtid, energy_eV );
        for( int k=0 ; k < nk ; k++) lookup[index+k] = ss[k] ;
    }
}





/**
_QPMT_mct_lpmtid
-----------------

* using templated payload size P as it needs to be a compile time constant
* parallelism over mct domain only
* loops over the provided list of pmtid


**/

#ifdef WITH_CUSTOM4
template <typename F, int P>
__global__ void _QPMT_mct_lpmtid(
    qpmt<F>* pmt,
    int etype,
    F* lookup ,
    const F* domain,
    unsigned domain_width,
    const int* lpmtid,
    unsigned num_lpmtid )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;

    //printf("//_QPMT_mct_lpmtid etype %d ix %d num_lpmtid %d P %d \n", etype, ix, num_lpmtid, P );

    F minus_cos_theta = domain[ix] ;
    F wavelength_nm = 440.f ;
    F dot_pol_cross_mom_nrm = 0.f ; // SPOL zero is pure P polarized
    F lposcost = 0.5f ;  // np.acos(0.5) 1.047197

    const int& ni = num_lpmtid ;
    const int& nj = domain_width ;   // minus_cos_theta values "AOI"
    const int&  j = ix ;

    F payload[P] ;

    for(int i=0 ; i < ni ; i++)  // over num_lpmtid
    {
        int pmtid = lpmtid[i] ;

        if( etype == qpmt_SPEC )
        {
            pmt->get_lpmtid_SPEC(payload, pmtid, wavelength_nm );
        }
        else if( etype == qpmt_SPEC_ce )
        {
            pmt->get_lpmtid_SPEC_ce(payload, pmtid, wavelength_nm, lposcost );
        }
        else if( etype == qpmt_LL )
        {
            pmt->get_lpmtid_LL(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm );
        }
        else if( etype == qpmt_COMP )
        {
            pmt->get_lpmtid_COMP(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm );
        }
        else if( etype == qpmt_ART )
        {
            pmt->get_lpmtid_ART(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm );
        }
        else if( etype == qpmt_ARTE )
        {
            pmt->get_lpmtid_ARTE(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm );
        }
        else if( etype == qpmt_ATQC )
        {
            pmt->get_lpmtid_ATQC(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm, lposcost );
        }


        int index = i*nj*P + j*P  ;  // output index
        for( int k=0 ; k < P ; k++) lookup[index+k] = payload[k] ;
    }
}


template <typename F> extern void QPMT_mct_lpmtid_scan(
    dim3 numBlocks,
    dim3 threadsPerBlock,
    qpmt<F>* pmt,
    int etype,
    F* lookup,
    const F* domain,
    unsigned domain_width,
    const int* lpmtid,
    unsigned num_lpmtid
)
{
    printf("//QPMT_mct_lpmtid_scan etype %d domain_width %d num_lpmtid %d \n", etype, domain_width, num_lpmtid);

    switch(etype)
    {
        case qpmt_SPEC:
           _QPMT_mct_lpmtid<F,16><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_SPEC_ce:
           _QPMT_mct_lpmtid<F,16><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_ART:
           _QPMT_mct_lpmtid<F,16><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_COMP:
           _QPMT_mct_lpmtid<F,32><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_LL:
           _QPMT_mct_lpmtid<F,128><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_ARTE:
           _QPMT_mct_lpmtid<F,4><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_ATQC:
           _QPMT_mct_lpmtid<F,4><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        default:
              printf("//PMT_mct_lpmtid_scan etype %d UNHANDLED \n", etype)   ; break ;

    }
}

template void QPMT_mct_lpmtid_scan<float>(   dim3, dim3, qpmt<float>*, int etype, float*,  const float* , unsigned, const int*, unsigned);
// end WITH_CUSTOM4
#endif








template <typename F>
__global__ void _QPMT_spmtid(
    qpmt<F>* pmt,
    int etype,
    F* lookup ,
    const int* spmtid,
    unsigned num_spmtid )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= num_spmtid ) return;
    int _spmtid = spmtid[ix];
    //printf("//_QPMT_spmtid etype %d ix %d num_spmtid %d _spmtid %d \n", etype, ix, num_spmtid, _spmtid );

    F value = 0.f ;
    if( etype == qpmt_S_QESCALE )
    {
        value = pmt->get_s_qescale_from_spmtid( _spmtid );
    }
    lookup[ix] = value ;
}




template <typename F> extern void QPMT_spmtid_scan(
    dim3 numBlocks,
    dim3 threadsPerBlock,
    qpmt<F>* pmt,
    int etype,
    F* lookup,
    const int* spmtid,
    unsigned num_spmtid
)
{
    printf("//QPMT_spmtid_scan etype %d num_spmtid %d \n", etype, num_spmtid);
    switch(etype)
    {
        case qpmt_S_QESCALE:
           _QPMT_spmtid<F><<<numBlocks,threadsPerBlock>>>(pmt, etype, lookup, spmtid, num_spmtid ) ;  break ;
    }
}

template void QPMT_spmtid_scan<float>( dim3, dim3, qpmt<float>*, int, float*, const int*, unsigned );


